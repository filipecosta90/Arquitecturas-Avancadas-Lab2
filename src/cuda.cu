/**************************************************************
 * 
 * --== Simple CUDA kernel ==--
 * author: ampereira
 * 
 *
 * Fill the rest of the code
 *
 * Insert the functions for time measurement in the correct 
 * sections (i.e. do not account for filling the vectors with random data)
 *
 * Before compile choose the CPU/CUDA version by running the bash command:
 *     export CUDA=yes    or    export CUDA=no
 *
 **************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>

#define TIME_RESOLUTION 1000000	// time measuring resolution (us)
#define NUM_BLOCKS 128
#define STRIDE_SIZE 16
#define NUM_THREADS_PER_BLOCK 256
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK
#define RADIUS 2

using namespace std;
timeval t;

long long unsigned cpu_time;
hipEvent_t start, stop;

// These are specific to measure the execution of only the kernel execution - might be useful
void startKernelTime (void) {
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
}

void stopKernelTime (char * discription) {
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  cout << milliseconds << " ms have elapsed for the kernel " << discription << " execution" << endl;
}

// Fill the input parameters and kernel qualifier
__global__ void stencilKernelStride (float *in, float *out) {

  for ( int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < SIZE; tid += STRIDE_SIZE ){
    float value = 0.0f;
    for ( int pos = -RADIUS; pos <= RADIUS; pos++ ){
      value += in[tid+pos];
    }
    out[tid]=value;
  }
}

__global__ void stencilKernelSharedMemory (float *in, float *out){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  float value = 0.0f;
  __shared__ float temp[NUM_THREADS_PER_BLOCK+2*RADIUS];
 temp[threadIdx.x + RADIUS] = in[tid];
	if(threadIdx.x < RADIUS ){
	 	//before
		if (tid - threadIdx.x >= 0){
			temp[threadIdx.x] = in[tid - threadIdx.x];
		}
	 	//after
		if ( tid + threadIdx.x < SIZE){
			temp[threadIdx.x+blockDim.x] = in[tid + threadIdx.x];
		}
	 }
  __syncthreads();

    for(int pos = 0; pos<=2*RADIUS; pos++){
    value += temp[pos];
  }

  out[tid] = value;
}

/*
// Fill the input parameters and kernel qualifier
void quicksortKernel (???) {

}
 */

// Fill with the code required for the GPU stencil (mem allocation, transfers, kernel launch....)
void stencilGPU (void) {

  int bytes = SIZE*sizeof(int);
  float vector[SIZE], output_vector[SIZE];
  float *dev_vector, *dev_output;

  // create random vector
  for (unsigned i = 0; i<SIZE; i++){
    vector[i]=(float) rand()/RAND_MAX;
  }

  // malloc memmory device
  hipMalloc((void**)&dev_vector,bytes);
  hipMalloc((void**)&dev_output,bytes);
  startKernelTime();
  // copy inputs to the device
  hipMemcpy(dev_vector,&vector,bytes,hipMemcpyHostToDevice);

  // launch the kernel
  // instead of number o blocks we now have stride size
  dim3 dimGrid(NUM_BLOCKS);
  dim3 dimBlock(NUM_THREADS_PER_BLOCK);

  stencilKernelStride<<<dimBlock,dimGrid>>>(dev_vector,dev_output);
  hipDeviceSynchronize();
  stopKernelTime("Stride");
  // copy the output to the host
  startKernelTime();
  stencilKernelSharedMemory <<<dimBlock,dimGrid>>>(dev_vector,dev_output);
  stopKernelTime("Shared Memory");
  startKernelTime();
  hipMemcpy(&output_vector,dev_output,bytes,hipMemcpyDeviceToHost);
  stopKernelTime("hipMemcpy");

  // free the device memory
  hipFree(dev_vector);
  hipFree(dev_output);
}
/*
// Fill with the code required for the GPU quicksort (mem allocation, transfers, kernel launch....)
void quicksortGPU (void) {

}*/

int main (int argc, char** argv){

  stencilGPU();
  return 0;
}

